#include "hip/hip_runtime.h"
#include <cosmictiger/options.hpp>
#include <cosmictiger/cuda_export.hpp>
#include <cosmictiger/cuda_check.hpp>
#include <cosmictiger/gravity_cuda.hpp>
#include <cosmictiger/green.hpp>
#include <cosmictiger/interactions.hpp>

#include <stack>
#include <atomic>
void yield_to_hpx();

__device__ __constant__ cuda_ewald_const cuda_ewald;

__device__ const cuda_ewald_const& cuda_get_const() {
	return cuda_ewald;
}

double *flop_ptr;

double cuda_reset_flop() {
	double result;
	double zero = 0.0;
	CUDA_CHECK(hipMemcpy(&result, flop_ptr, sizeof(double), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(flop_ptr, &zero, sizeof(double), hipMemcpyHostToDevice));
	return result;
}

void cuda_init() {
	static std::atomic<int> lock(0);
	static bool init = false;
	while (lock++ != 0) {
		lock--;
	}
	if (!init) {
		static const float efs[LP + 1] = { 1.00000000e+00, 1.00000000e+00, 1.00000000e+00, 1.00000000e+00, 5.00000000e-01, 1.00000000e+00, 1.00000000e+00,
				5.00000000e-01, 1.00000000e+00, 5.00000000e-01, 1.66666667e-01, 5.00000000e-01, 5.00000000e-01, 5.00000000e-01, 1.00000000e+00, 5.00000000e-01,
				1.66666667e-01, 5.00000000e-01, 5.00000000e-01, 1.66666667e-01, 4.16666667e-02, 1.66666667e-01, 1.66666667e-01, 2.50000000e-01, 5.00000000e-01,
				2.50000000e-01, 1.66666667e-01, 5.00000000e-01, 5.00000000e-01, 1.66666667e-01, 4.16666667e-02, 1.66666667e-01, 2.50000000e-01, 1.66666667e-01,
				4.16666667e-02, 0.0 };

		static cuda_ewald_const c;
		const ewald_indices indices_real(EWALD_REAL_N2, false);
		const ewald_indices indices_four(EWALD_FOUR_N2, true);
		const periodic_parts periodic;
		for (int i = 0; i < indices_real.size(); i++) {
			c.real_indices[i] = indices_real[i];
		}
		for (int i = 0; i < indices_four.size(); i++) {
			c.four_indices[i] = indices_four[i];
			c.periodic_parts[i] = periodic[i];
		}
		for (int i = 0; i < LP; i++) {
			c.exp_factors[i] = efs[i];
		}

		CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cuda_ewald), &c, sizeof(cuda_ewald_const)));
		init = true;
		CUDA_CHECK(hipHostMalloc((void** )&flop_ptr, sizeof(double)));
		cuda_reset_flop();
//		CUDA_CHECK(cudaThreadSetLimit(hipLimitStackSize, 2048));
	}
	lock--;
}

#define WARPSIZE 32
#define CCSIZE 32

#define WORKSIZE 256
#define PCWORKSIZE 96
#define NODESIZE 64
#define NWARP (WORKSIZE/WARPSIZE)
#define PCNWARP (PCWORKSIZE/WARPSIZE)
#define WARPSIZE 32

#include <cstdint>

#define TILESIZE 512

__global__ void gravity_ewald_direct_kernel(_4force *f, vect<double> *x, vect<double> *y, int xsize, int ysize, float m, float h) {
	const int i = threadIdx.x;
	const auto &cuda_const = cuda_get_const();
	const auto &four_indices = cuda_const.four_indices;
	const auto &real_indices = cuda_const.real_indices;
	const auto &hparts = cuda_const.periodic_parts;
	const float fouroversqrtpi(4.0 / sqrt(M_PI));
	static const float one(1.0);
	static const float two(2.0);
	static const float nfour(-4.0);
	static const float a1(0.254829592);
	static const float a2(-0.284496736);
	static const float a3(1.421413741);
	static const float a4(-1.453152027);
	static const float a5(1.061405429);
	static const float rcut(1.0e-6);
	static const float twopi = 2.0 * M_PI;
	static const float p(0.3275911);
	const float hinv = 1.0 / h;
	const float h3inv = hinv * hinv * hinv;

	for (int j = 0; j < xsize; j += TILESIZE) {
		const int k = j + i;
		if (k < xsize) {
			f[k].phi = m * SELF_PHI / h;
			f[k].g = vect<float>(0.0);
			for (int l = 0; l < ysize; l++) {
				vect<float> X;
				for (int dim = 0; dim < NDIM; dim++) {
					const auto dx = x[k][dim] - y[l][dim];
					X[dim] = float(copysign(min(abs(dx),double(1.0) - abs(dx)), dx * (double(0.5) - abs(dx))));
				}
				const float r = abs(X);
				if (r > h) {
					_4force freal;
					_4force ffour;
					freal.g = ffour.g = vect<float>(0.0);
					freal.phi = ffour.phi = 0.0;
//					for (auto n : real_indices) {
//						const vect<float> dx = X - vect<float>(n);				// 3
//						const float r2 = dx.dot(dx);							// 5
//						if (r2 < (EWALD_RADIUS_CUTOFF * EWALD_RADIUS_CUTOFF)) {	// 1
//							const float r = sqrt(r2);					// 1
//							const float cmask = one - (n.dot(n) > 0.0); // 7
//							const float rinv = one / r;		// 2
//							const float r2inv = rinv * rinv;			// 1
//							const float r3inv = r2inv * rinv;			// 1
//							const float t1 = float(1) / (float(1) + p * two * r); 	//4
//							const float t2 = t1 * t1;								// 1
//							const float t3 = t2 * t1;								// 1
//							const float t4 = t2 * t2;								// 1
//							const float t5 = t2 * t3;								// 1
//							const float exp0 = expf(nfour * r2);					// 26
//							const float erfc0 = (a1 * t1 + a2 * t2 + a3 * t3 + a4 * t4 + a5 * t5) * exp0; 			// 10
//							const float expfactor = fouroversqrtpi * r * exp0; 	// 2
//							const float e1 = expfactor * r3inv;						// 1
//							const float d0 = -erfc0 * rinv;							// 2
//							const float d1 = fma(-d0, r2inv, e1);					// 3
//							freal.phi += d0;
//							freal.g -= dx * d1;
//						}
//					}
//					for (int n = 0; n < EWALD_NFOUR; n++) {
//						const auto &h = four_indices[n];
//						const auto &hpart = hparts[n];
//						const float h2 = h.dot(h);
//						const float hdotx = h.dot(X);
//						float co;
//						float so;
//						sincosf(twopi * hdotx, &so, &co);
//						ffour.phi += hpart() * co;
//						for (int dim = 0; dim < NDIM; dim++) {
//							ffour.g[dim] -= hpart(dim) * so;
//						}
//					}
					f[k].phi -= m / r;
					f[k].g -= X * m / (r * r * r);
//					f[k].phi += (ffour.phi + freal.phi) * m;
					//					f[k].g += (ffour.g + freal.g) * m;
				} else {
					const float rinv = 1.0 / r;
					const float rinv3 = rinv * rinv;
					float p, f0;
					if (r > 0.5 * h) {
						const float roh = min(r * hinv, 1.0);                         // 2
						const float roh2 = roh * roh;                         // 1
						const float roh3 = roh2 * roh;                         // 1
						f0 = float(-32.0 / 3.0);
						f0 = fma(f0, roh, float(+192.0 / 5.0));                         // 2
						f0 = fma(f0, roh, float(-48.0));                         // 2
						f0 = fma(f0, roh, float(+64.0 / 3.0));                         // 2
						f0 = fma(f0, roh3, float(-1.0 / 15.0));                         // 2
						f0 *= rinv3;                         // 1
						p = float(+32.0 / 15.0);
						p = fma(p, roh, float(-48.0 / 5.0));                                 // 2
						p = fma(p, roh, float(+16.0));                                 // 2
						p = fma(p, roh, float(-32.0 / 3.0));                                 // 2
						p = fma(p, roh2, float(+16.0 / 5.0));                                 // 2
						p = fma(p, roh, float(-1.0 / 15.0));                                 // 2
						p *= rinv;                                 // 1
					} else {
						const float roh = min(r * hinv, 1.0);                           // 2
						const float roh2 = roh * roh;                           // 1
						f0 = float(+32.0);
						f0 = fma(f0, roh, float(-192.0 / 5.0));                           // 2
						f0 = fma(f0, roh2, float(+32.0 / 3.0));                           // 2
						f0 *= h3inv;                           // 1
						p = float(-32.0 / 5.0);
						p = fma(p, roh, float(+48.0 / 5.0));							// 2
						p = fma(p, roh2, float(-16.0 / 3.0));							// 2
						p = fma(p, roh2, float(+14.0 / 5.0));							// 2
						p *= hinv;							// 1
					}
					const auto dXM = X * m;								// 3
					f[k].g -= dXM * f0;
					f[k].phi -= p * m;
				}
			}
		}
	}
}

void gravity_ewald_direct(std::vector<_4force> &f, const std::vector<vect<double>> x, const std::vector<vect<double>> &y) {
	_4force *fdev;
	vect<double> *xdev;
	vect<double> *ydev;
	const auto xbytes = x.size() * sizeof(vect<double> );
	const auto ybytes = y.size() * sizeof(vect<double> );
	const auto fbytes = f.size() * sizeof(_4force);
	CUDA_CHECK(hipMalloc((void** ) &fdev, fbytes));
	CUDA_CHECK(hipMalloc((void** ) &xdev, xbytes));
	CUDA_CHECK(hipMalloc((void** ) &ydev, ybytes));
	CUDA_CHECK(hipMemcpy(xdev, x.data(), xbytes, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(ydev, y.data(), ybytes, hipMemcpyHostToDevice));
	/**/gravity_ewald_direct_kernel<<<1,TILESIZE>>>(fdev, xdev, ydev, x.size(), y.size(), opts.particle_mass, opts.h);

	CUDA_CHECK(hipMemcpy(f.data(), fdev, fbytes, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipFree(fdev));
	CUDA_CHECK(hipFree(xdev));
	CUDA_CHECK(hipFree(ydev));
}

__global__ void CC_ewald_kernel(expansion<float> *lptr, const vect<position> X, const multi_src *y, int ysize, bool do_phi, double *flop_ptr) {

	int l = threadIdx.x + blockDim.x * blockIdx.x;
	int n = threadIdx.x;
	int tb_size = blockDim.x;
	auto &L = *lptr;

	__shared__ expansion<float>
	Lacc[CCSIZE];
	__shared__ std::uint64_t
	flop[CCSIZE];
	flop[n] = 0;
	for (int i = 0; i < LP; i++) {
		Lacc[n][i] = 0.0;
	}
	for (int yi = l; yi < ysize; yi += tb_size * gridDim.x) {
		if (yi < ysize) {
			vect<float> dX;
			for (int dim = 0; dim < NDIM; dim++) {
				dX[dim] = float(int(X[dim]) - int(y[yi].x[dim])) * float(POS_INV); 		// 3
			}
			flop[n] += 3 + multipole_interaction(Lacc[n], y[yi].m, dX, true, do_phi);
		}
	}
	for (int N = tb_size / 2; N > 0; N >>= 1) {
		if (n < N) {
			for (int i = 0; i < LP; i++) {
				Lacc[n][i] += Lacc[n + N][i];
			}
			flop[n] += LP;
		}
	}
	if (n == 0) {
		for (int i = 0; i < LP; i++) {
			atomicAdd(&L[i], Lacc[0][i]);
		}
		flop[n] += LP;
	}
	for (int N = tb_size / 2; N > 0; N >>= 1) {
		if (n < N) {
			flop[n] += flop[n + N];
		}
	}
	if (n == 0) {
		atomicAdd(flop_ptr, flop[0]);
	}
}

struct cuda_context_ewald {
	int ysize;
	hipStream_t stream;
	expansion<float> *L;
	multi_src *y;
	expansion<float> *Lp;
	multi_src *yp;
	cuda_context_ewald(int ys) {
		ysize = 1;
		while (ysize < ys) {
			ysize *= 2;
		}
		CUDA_CHECK(hipMalloc(&L, sizeof(expansion<float> )));
		CUDA_CHECK(hipMalloc(&y, sizeof(multi_src) * ysize));
		CUDA_CHECK(hipHostMalloc(&Lp, sizeof(expansion<float> )));
		CUDA_CHECK(hipHostMalloc(&yp, sizeof(multi_src) * ysize));
		CUDA_CHECK(hipStreamCreate(&stream));
	}
	void resize(int ys) {
		if (ys > ysize) {
			while (ysize < ys) {
				ysize *= 2;
			}
			CUDA_CHECK(hipFree(y));
			CUDA_CHECK(hipMalloc(&y, sizeof(multi_src) * ysize));
			CUDA_CHECK(hipHostFree(yp));
			CUDA_CHECK(hipHostMalloc(&yp, sizeof(multi_src) * ysize));
		}
	}
};

static std::atomic<int> lock_ewald(0);
static std::stack<cuda_context_ewald> stack_ewald;

cuda_context_ewald pop_context_ewald(int ys) {
	while (lock_ewald++ != 0) {
		lock_ewald--;
	}
	if (stack_ewald.empty()) {
		lock_ewald--;
		return cuda_context_ewald(ys);
	} else {
		auto ctx = stack_ewald.top();
		stack_ewald.pop();
		lock_ewald--;
		ctx.resize(ys);
		return ctx;
	}
}

void push_context_ewald(cuda_context_ewald ctx) {
	while (lock_ewald++ != 0) {
		lock_ewald--;
	}
	stack_ewald.push(ctx);
	lock_ewald--;
}

void gravity_CC_ewald_cuda(expansion<float> &L, const vect<position> &x, std::vector<const multi_src*> &y, bool do_phi) {

	cuda_init();

	auto ctx = pop_context_ewald(y.size());
	int k = 0;
	for (int i = 0; i < y.size(); i++) {
		ctx.yp[k++] = *y[i];
	}
	*ctx.Lp = L;
	CUDA_CHECK(hipMemcpyAsync(ctx.y, ctx.yp, sizeof(multi_src) * y.size(), hipMemcpyHostToDevice, ctx.stream));
	CUDA_CHECK(hipMemcpyAsync(ctx.L, ctx.Lp, sizeof(expansion<float> ), hipMemcpyHostToDevice, ctx.stream));

	int tb_size = (((y.size() - 1) / CCSIZE) + 1) * CCSIZE;

	/**/CC_ewald_kernel<<<dim3(tb_size/CCSIZE,1,1),dim3(CCSIZE,1,1),0,ctx.stream>>>(ctx.L, x, ctx.y, y.size(), do_phi, flop_ptr);

	CUDA_CHECK(hipMemcpyAsync(ctx.Lp, ctx.L, sizeof(expansion<float> ), hipMemcpyDeviceToHost, ctx.stream));
	while (hipStreamQuery(ctx.stream) != hipSuccess) {
		yield_to_hpx();
	}
	L = *ctx.Lp;
	push_context_ewald(std::move(ctx));
}

template<bool DO_PHI>
/**/__global__ /**/
void PP_direct_kernel(_4force *F, const vect<position> *x, const vect<position> *y, const std::pair<int, int> *yiters, int *xindex,
		int *yindex, float m, float h, double *flop_ptr) {
//	printf("sizeof(_4force) = %li\n", sizeof(_4force));

	const int iwarp = threadIdx.y;
	const int ui = blockIdx.x;
	const int l = iwarp * blockDim.x + threadIdx.x;
	const int n = threadIdx.x;
	const float Hinv = 1.0 / h;
	const float H3inv = Hinv * Hinv * Hinv;
	const float halfh = 0.5 * h;

	__shared__ vect<position>
	X[NODESIZE];
	__shared__ _4force
	G[NWARP][WARPSIZE];
	__shared__ vect<position>
	Ymem[NWARP][WARPSIZE][SYNCRATE];

	__shared__ std::uint64_t
	flop[NWARP][WARPSIZE];

	flop[iwarp][n] = 0;

	const auto yb = yindex[ui];
	const auto ye = yindex[ui + 1];
	const auto xb = xindex[ui];
	const auto xe = xindex[ui + 1];
	const auto xsize = xe - xb;
	if (l < xsize) {
		X[l] = x[xb + l];
	}
	__syncthreads();
	{
		const auto ymax = ((ye - yb - 1) / WORKSIZE + 1) * WORKSIZE + yb;
		for (int yi = yb + l; yi < ymax; yi += WORKSIZE) {
			int jb, je;
			if (yi < ye) {
				jb = yiters[yi].first;
				je = yiters[yi].second;
			}
			for (int k = 0; k < WARPSIZE; k++) {
				auto *Yptr = reinterpret_cast<float*>(Ymem[iwarp][k]);
				const int this_yi = ((yi - yb) / WARPSIZE) * WARPSIZE + k + yb;
				if (this_yi < ye) {
					const int jb = yiters[this_yi].first;
					const int je = yiters[this_yi].second;
					const int size = (je - jb) * NDIM;
					if (n < size) {
						Yptr[n] = reinterpret_cast<const float*>(y + jb)[n];
					}
				}
			}
			for (int i = xb; i < xe; i++) {
				const auto this_x = X[i - xb];
				G[iwarp][n].phi = 0.0;
				G[iwarp][n].g = vect<float>(0.0);
				if (yi < ye) {
#pragma loop unroll SYNCRATE
					for (int j0 = 0; j0 < SYNCRATE; j0++) {
						const int j = j0 + jb;
						if (j < je) {
							const vect<position> Y = Ymem[iwarp][n][j0];
							vect<float> dX;
							for (int dim = 0; dim < NDIM; dim++) {
								dX[dim] = float(int(this_x[dim]) - int(Y[dim])) * float(POS_INV);			// 3
							}
							flop[iwarp][n] += 3;
							const float r2 = dX.dot(dX);								   // 5
							const float r = sqrt(r2);// 1
							const float rinv = float(1) / max(r, halfh);// 2
							const float rinv3 = rinv * rinv * rinv;// 2
							flop[iwarp][n] += DO_PHI ? 21 : 19;
							float f, p;
							if (r > h) {
								f = rinv3;
								p = rinv;
							} else if (r > 0.5 * h) {
								const float roh = min(r * Hinv, 1.0);                         // 2
								const float roh2 = roh * roh;// 1
								const float roh3 = roh2 * roh;// 1
								f = float(-32.0 / 3.0);
								f = fma(f, roh, float(+192.0 / 5.0));// 2
								f = fma(f, roh, float(-48.0));// 2
								f = fma(f, roh, float(+64.0 / 3.0));// 2
								f = fma(f, roh3, float(-1.0 / 15.0));// 2
								f *= rinv3;// 1
								flop[iwarp][n] += 13;
								if (DO_PHI) {
									p = float(+32.0 / 15.0);
									p = fma(p, roh, float(-48.0 / 5.0));                                 // 2
									p = fma(p, roh, float(+16.0));// 2
									p = fma(p, roh, float(-32.0 / 3.0));// 2
									p = fma(p, roh2, float(+16.0 / 5.0));// 2
									p = fma(p, roh, float(-1.0 / 15.0));// 2
									p *= rinv;// 1
									flop[iwarp][n] += 11;
								}
							} else {
								const float roh = min(r * Hinv, 1.0);                           // 2
								const float roh2 = roh * roh;// 1
								f = float(+32.0);
								f = fma(f, roh, float(-192.0 / 5.0));// 2
								f = fma(f, roh2, float(+32.0 / 3.0));// 2
								f *= H3inv;// 1
								flop[iwarp][n] += 7;
								if (DO_PHI) {
									p = float(-32.0 / 5.0);
									p = fma(p, roh, float(+48.0 / 5.0));							// 2
									p = fma(p, roh2, float(-16.0 / 3.0));// 2
									p = fma(p, roh2, float(+14.0 / 5.0));// 2
									p *= Hinv;// 1
									flop[iwarp][n] += 8;
								}
							}
							const auto dXM = dX * m;								// 3
							for (int dim = 0; dim < NDIM; dim++) {
								G[iwarp][n].g[dim] -= dXM[dim] * f;    				// 6
							}
							// 13S + 2D = 15
							if( DO_PHI ) {
								G[iwarp][n].phi -= p * m;    							// 2
							}
						}
					}
				}
				for (int N = WARPSIZE / 2; N > 0; N >>= 1) {
					if (n < N) {
						G[iwarp][n].g += G[iwarp][n + N].g;
						flop[iwarp][n] += 4;
						if( DO_PHI ) {
							G[iwarp][n].phi += G[iwarp][n + N].phi;
							flop[iwarp][n] += 1;
						}
					}
				}
				if (n == 0) {
					for (int dim = 0; dim < NDIM; dim++) {
						atomicAdd(&F[i].g[dim], G[iwarp][0].g[dim]);
					}
					flop[iwarp][n] += 4;
					if( DO_PHI ) {
						atomicAdd(&F[i].phi, G[iwarp][0].phi);
						flop[iwarp][n] += 1;
					}
				}
			}
		}
	}
	for (int N = WARPSIZE / 2; N > 0; N >>= 1) {
		if (n < N) {
			flop[iwarp][n] += flop[iwarp][n + N];
		}
	}
	if (n == 0) {
		atomicAdd(flop_ptr, flop[iwarp][0]);
	}
}

__global__
void PC_direct_kernel(_4force *F, const vect<position> *x, const multi_src *z, int *xindex, int *zindex, bool do_phi, double *flop_ptr) {
//	printf("sizeof(_4force) = %li\n", sizeof(_4force));

	const int iwarp = threadIdx.y;
	const int ui = blockIdx.x;
	const int l = iwarp * blockDim.x + threadIdx.x;
	const int n = threadIdx.x;

	__shared__ vect<position>
	X[NODESIZE];
	__shared__ _4force
	G[PCNWARP][WARPSIZE];
	__shared__ std::uint64_t
	flop[NWARP][WARPSIZE];

	flop[iwarp][n] = 0;

	const auto xb = xindex[ui];
	const auto xe = xindex[ui + 1];
	const auto xsize = xe - xb;
	if (l < xsize) {
		X[l] = x[xb + l];
	}
	__syncthreads();
	int zmax = ((zindex[ui + 1] - zindex[ui] - 1) / PCWORKSIZE + 1) * PCWORKSIZE + zindex[ui];
	for (int zi = zindex[ui] + l; zi < zmax; zi += PCWORKSIZE) {
		for (int i = xb; i < xe; i++) {
			G[iwarp][n].phi = 0.0;
			G[iwarp][n].g = vect<float>(0.0);
			if (zi < zindex[ui + 1]) {
				const multipole<float> &M = z[zi].m;
				const vect<position> &Y = z[zi].x;
				vect<float> dX;
				for (int dim = 0; dim < NDIM; dim++) {
					dX[dim] = float(int(X[i - xb][dim]) - int(Y[dim])) * float(POS_INV); // 3
				}
				flop[iwarp][n] += 3;

				vect<float> g;
				float phi;
				flop[iwarp][n] += 4 + multipole_interaction(g, phi, M, dX, false, do_phi); // 516
				G[iwarp][n].g += g; // 0 / 3
				G[iwarp][n].phi += phi; // 0 / 1
			}
			for (int N = WARPSIZE / 2; N > 0; N >>= 1) {
				if (n < N) {
					G[iwarp][n].g += G[iwarp][n + N].g;
					G[iwarp][n].phi += G[iwarp][n + N].phi;
					flop[iwarp][n] += 4;
				}
			}
			if (n == 0) {
				for (int dim = 0; dim < NDIM; dim++) {
					atomicAdd(&F[i].g[dim], G[iwarp][0].g[dim]);
				}
				atomicAdd(&F[i].phi, G[iwarp][0].phi);
				flop[iwarp][0] += 4;
			}
		}
	}
	for (int N = WARPSIZE / 2; N > 0; N >>= 1) {
		if (n < N) {
			flop[iwarp][n] += flop[iwarp][n + N];
		}
	}
	if (n == 0) {
		atomicAdd(flop_ptr, flop[iwarp][0]);
	}
}

struct cuda_context {
	int xsize, ysize, zsize, isize, ypsize;
	hipStream_t stream;
	_4force *f;
	vect<position> *x;
	std::pair<int, int> *y;
	vect<position> *ypos;
	multi_src *z;
	int *xi;
	int *yi;
	int *zi;
	_4force *fp;
	vect<position> *xp;
	multi_src *zp;
	std::pair<int, int> *yp;
	int *xip;
	int *yip;
	int *zip;
	cuda_context(int xs, int ys, int zs, int is, int yps) {
		xsize = 1;
		ysize = 1;
		zsize = 1;
		isize = 1;
		ypsize = 1;
		while (xsize < xs) {
			xsize *= 2;
		}
		while (zsize < zs) {
			zsize *= 2;
		}
		while (ysize < ys) {
			ysize *= 2;
		}
		while (ypsize < yps) {
			ypsize *= 2;
		}
		while (isize < is) {
			isize *= 2;
		}
		CUDA_CHECK(hipMalloc(&f, sizeof(_4force) * xsize));
		CUDA_CHECK(hipMalloc(&x, sizeof(vect<position> ) * xsize));
		CUDA_CHECK(hipMalloc(&y, sizeof(std::pair<int, int>) * ysize));
		CUDA_CHECK(hipMalloc(&ypos, sizeof(vect<position> ) * ypsize));
		CUDA_CHECK(hipMalloc(&z, sizeof(multi_src) * zsize));
		CUDA_CHECK(hipMalloc(&xi, sizeof(int) * isize));
		CUDA_CHECK(hipMalloc(&yi, sizeof(int) * isize));
		CUDA_CHECK(hipMalloc(&zi, sizeof(int) * isize));
		CUDA_CHECK(hipHostMalloc(&fp, sizeof(_4force) * xsize));
		CUDA_CHECK(hipHostMalloc(&xp, sizeof(vect<position> ) * xsize));
		CUDA_CHECK(hipHostMalloc(&yp, sizeof(std::pair<int, int>) * ysize));
		CUDA_CHECK(hipHostMalloc(&zp, sizeof(multi_src) * zsize));
		CUDA_CHECK(hipHostMalloc(&xip, sizeof(int) * isize));
		CUDA_CHECK(hipHostMalloc(&yip, sizeof(int) * isize));
		CUDA_CHECK(hipHostMalloc(&zip, sizeof(int) * isize));
		CUDA_CHECK(hipStreamCreate(&stream));
	}
	void resize(int xs, int ys, int zs, int is, int yps) {
		if (yps > ypsize) {
			while (ypsize < yps) {
				ypsize *= 2;
			}
			CUDA_CHECK(hipFree(ypos));
			CUDA_CHECK(hipMalloc(&ypos, sizeof(vect<position> ) * ypsize));
		}
		if (xs > xsize) {
			while (xsize < xs) {
				xsize *= 2;
			}
			CUDA_CHECK(hipFree(x));
			CUDA_CHECK(hipFree(f));
			CUDA_CHECK(hipMalloc(&f, sizeof(_4force) * xsize));
			CUDA_CHECK(hipMalloc(&x, sizeof(vect<position> ) * xsize));
			CUDA_CHECK(hipHostFree(xp));
			CUDA_CHECK(hipHostFree(fp));
			CUDA_CHECK(hipHostMalloc(&fp, sizeof(_4force) * xsize));
			CUDA_CHECK(hipHostMalloc(&xp, sizeof(vect<position> ) * xsize));
		}
		if (ys > ysize) {
			while (ysize < ys) {
				ysize *= 2;
			}
			CUDA_CHECK(hipFree(y));
			CUDA_CHECK(hipMalloc(&y, sizeof(std::pair<int, int>) * ysize));
			CUDA_CHECK(hipHostFree(yp));
			CUDA_CHECK(hipHostMalloc(&yp, sizeof(std::pair<int, int>) * ysize));
		}
		if (zs > zsize) {
			while (zsize < zs) {
				zsize *= 2;
			}
			CUDA_CHECK(hipFree(z));
			CUDA_CHECK(hipMalloc(&z, sizeof(multi_src) * zsize));
			CUDA_CHECK(hipHostFree(zp));
			CUDA_CHECK(hipHostMalloc(&zp, sizeof(multi_src) * zsize));
		}
		if (is > isize) {
			while (isize < is) {
				isize *= 2;
			}
			CUDA_CHECK(hipFree(xi));
			CUDA_CHECK(hipFree(yi));
			CUDA_CHECK(hipFree(zi));
			CUDA_CHECK(hipMalloc(&xi, sizeof(int) * isize));
			CUDA_CHECK(hipMalloc(&yi, sizeof(int) * isize));
			CUDA_CHECK(hipMalloc(&zi, sizeof(int) * isize));
			CUDA_CHECK(hipHostFree(xip));
			CUDA_CHECK(hipHostFree(yip));
			CUDA_CHECK(hipHostFree(zip));
			CUDA_CHECK(hipHostMalloc(&xip, sizeof(int) * isize));
			CUDA_CHECK(hipHostMalloc(&yip, sizeof(int) * isize));
			CUDA_CHECK(hipHostMalloc(&zip, sizeof(int) * isize));
		}
	}
};

static std::atomic<int> lock(0);
static std::stack<cuda_context> stack;

cuda_context pop_context(int xs, int ys, int zs, int is, int yps) {
	while (lock++ != 0) {
		lock--;
	}
	if (stack.empty()) {
		lock--;
		return cuda_context(xs, ys, zs, is, yps);
	} else {
		auto ctx = stack.top();
		stack.pop();
		lock--;
		ctx.resize(xs, ys, zs, is, yps);
		return ctx;
	}
}

void push_context(cuda_context ctx) {
	while (lock++ != 0) {
		lock--;
	}
	stack.push(ctx);
	lock--;
}

struct pinned_context {
	pinned_vector<int> xindex;
	pinned_vector<int> yindex;
	pinned_vector<_4force> f;
	pinned_vector<vect<position>> x;
	pinned_vector<std::pair<int, int>> y;
	pinned_vector<multi_src> z;
	pinned_vector<int> zindex;
	pinned_context() = default;
	pinned_context(const pinned_context&) = delete;
	pinned_context(pinned_context&&) = default;
	pinned_context& operator=(const pinned_context&) = delete;
	pinned_context& operator=(pinned_context&&) = default;
};

std::stack<pinned_context> pinned_stack;
std::atomic<int> pinned_mtx(0);

pinned_context pop_pinned_context() {
	while (pinned_mtx++ != 0) {
		pinned_mtx--;
	}
	pinned_context ctx;
	if (!pinned_stack.empty()) {
		ctx = std::move(pinned_stack.top());
		pinned_stack.pop();
	}
	pinned_mtx--;
	return std::move(ctx);
}

void push_pinned_context(pinned_context &&ctx) {
	while (pinned_mtx++ != 0) {
		pinned_mtx--;
	}
	pinned_stack.push(std::move(ctx));
	pinned_mtx--;
}

void gravity_PP_direct_cuda(std::vector<cuda_work_unit> &&units, const pinned_vector<vect<position>> &ydata, bool do_phi) {
	static const float m = opts.particle_mass;
	cuda_init();
	std::uint64_t interactions = 0;
	{
		auto pctx = pop_pinned_context();
		auto &xindex = pctx.xindex;
		auto &yindex = pctx.yindex;
		auto &f = pctx.f;
		auto &x = pctx.x;
		auto &y = pctx.y;
		xindex.resize(0);
		yindex.resize(0);
		f.resize(0);
		x.resize(0);
		y.resize(0);
		int xi = 0;
		int yi = 0;
		for (const auto &unit : units) {
			xindex.push_back(xi);
			yindex.push_back(yi);
			xi += unit.xptr->size();
			yi += unit.yiters.size();
			for (const auto &this_f : *unit.fptr) {
				f.push_back(this_f);
			}
			for (const auto &this_x : *unit.xptr) {
				x.push_back(this_x);
			}
			for (int j = 0; j < unit.yiters.size(); j++) {
				std::pair<int, int> iter = unit.yiters[j];
				interactions += unit.xptr->size() * (iter.second - iter.first);
				y.push_back(iter);
			}
		}
		xindex.push_back(xi);
		yindex.push_back(yi);
		const auto fbytes = sizeof(_4force) * f.size();
		const auto xbytes = sizeof(vect<position> ) * x.size();
		const auto ybytes = sizeof(std::pair<int, int>) * y.size();
		const auto ypbytes = sizeof(vect<position> ) * ydata.size();
		const auto xibytes = sizeof(int) * xindex.size();
		const auto yibytes = sizeof(int) * yindex.size();

		auto ctx = pop_context(x.size(), y.size(), 0, xindex.size(), ydata.size());
		memcpy(ctx.fp, f.data(), fbytes);
		memcpy(ctx.xp, x.data(), xbytes);
		memcpy(ctx.yp, y.data(), ybytes);
		memcpy(ctx.xip, xindex.data(), xibytes);
		memcpy(ctx.yip, yindex.data(), yibytes);
		CUDA_CHECK(hipMemcpyAsync(ctx.f, ctx.fp, fbytes, hipMemcpyHostToDevice, ctx.stream));
		CUDA_CHECK(hipMemcpyAsync(ctx.y, y.data(), ybytes, hipMemcpyHostToDevice, ctx.stream));
		CUDA_CHECK(hipMemcpyAsync(ctx.ypos, ydata.data(), ypbytes, hipMemcpyHostToDevice, ctx.stream));
		CUDA_CHECK(hipMemcpyAsync(ctx.x, ctx.xp, xbytes, hipMemcpyHostToDevice, ctx.stream));
		CUDA_CHECK(hipMemcpyAsync(ctx.yi, yindex.data(), yibytes, hipMemcpyHostToDevice, ctx.stream));
		CUDA_CHECK(hipMemcpyAsync(ctx.xi, xindex.data(), xibytes, hipMemcpyHostToDevice, ctx.stream));
		if (do_phi) {
		PP_direct_kernel<true><<<dim3(units.size(),1,1),dim3(WARPSIZE,NWARP,1),0,ctx.stream>>>(ctx.f,ctx.x,ctx.ypos, ctx.y,ctx.xi,ctx.yi, m, opts.soft_len, flop_ptr);
	} else {
	PP_direct_kernel<false><<<dim3(units.size(),1,1),dim3(WARPSIZE,NWARP,1),0,ctx.stream>>>(ctx.f,ctx.x,ctx.ypos, ctx.y,ctx.xi,ctx.yi, m, opts.soft_len, flop_ptr);
}

CUDA_CHECK(hipMemcpyAsync(f.data(), ctx.f, fbytes, hipMemcpyDeviceToHost, ctx.stream));
while (hipStreamQuery(ctx.stream) != hipSuccess) {
	yield_to_hpx();
}
int k = 0;
for (const auto &unit : units) {
	for (auto &this_f : *unit.fptr) {
		this_f = f[k];
		k++;
	}
}
push_context(std::move(ctx));
push_pinned_context(std::move(pctx));

}
{
auto pctx = pop_pinned_context();
auto &xindex = pctx.xindex;
auto &z = pctx.z;
auto &zindex = pctx.zindex;
auto &f = pctx.f;
auto &x = pctx.x;
xindex.resize(0);
zindex.resize(0);
f.resize(0);
x.resize(0);
z.resize(0);

int xi = 0;
int zi = 0;
int size = 0;
std::uint64_t interactions = 0;
for (const auto &unit : units) {
	if (unit.z.size()) {
		xindex.push_back(xi);
		zindex.push_back(zi);
		xi += unit.xptr->size();
		zi += unit.z.size();
		for (const auto &this_f : *unit.fptr) {
			f.push_back(this_f);
		}
		for (const auto &this_x : *unit.xptr) {
			x.push_back(this_x);
		}
		for (int j = 0; j < unit.z.size(); j++) {
			z.push_back(*unit.z[j]);
		}
		size++;
	}
}
xindex.push_back(xi);
zindex.push_back(zi);
if (z.size()) {
	const auto fbytes = sizeof(_4force) * f.size();
	const auto xbytes = sizeof(vect<position> ) * x.size();
	const auto zbytes = sizeof(multi_src) * z.size();
	const auto xibytes = sizeof(int) * xindex.size();
	const auto zibytes = sizeof(int) * zindex.size();

	auto ctx = pop_context(x.size(), 0, z.size(), zindex.size(), 0);
	CUDA_CHECK(hipMemcpyAsync(ctx.f, f.data(), fbytes, hipMemcpyHostToDevice, ctx.stream));
//		printf( "%li %lli %lli\n", zbytes, ctx.z, ctx.zp);
	CUDA_CHECK(hipMemcpyAsync(ctx.z, z.data(), zbytes, hipMemcpyHostToDevice, ctx.stream));
	CUDA_CHECK(hipMemcpyAsync(ctx.x, x.data(), xbytes, hipMemcpyHostToDevice, ctx.stream));
	CUDA_CHECK(hipMemcpyAsync(ctx.xi, xindex.data(), xibytes, hipMemcpyHostToDevice, ctx.stream));
	CUDA_CHECK(hipMemcpyAsync(ctx.zi, zindex.data(), zibytes, hipMemcpyHostToDevice, ctx.stream));

	/**/PC_direct_kernel<<<dim3(size,1,1),dim3(WARPSIZE,PCNWARP,1),0,ctx.stream>>>(ctx.f,ctx.x,ctx.z,ctx.xi,ctx.zi, do_phi, flop_ptr);

			CUDA_CHECK(hipMemcpyAsync(f.data(), ctx.f, fbytes, hipMemcpyDeviceToHost, ctx.stream));
	while (hipStreamQuery(ctx.stream) != hipSuccess) {
		yield_to_hpx();
	}
	int k = 0;
	for (const auto &unit : units) {
		if (unit.z.size()) {
			for (auto &this_f : *unit.fptr) {
				this_f = f[k];
				k++;
			}
		}
	}
	push_context(ctx);
}
push_pinned_context(std::move(pctx));
}
}

